#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>         // CUtensormap
#include <cuda/barrier>
using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

#define SMEM_WIDTH 64
#define SMEM_HEIGHT 64

__global__ void kernel(const __grid_constant__ CUtensorMap tensor_map, int x, int y) {
  // The destination shared memory buffer of a bulk tensor operation should be
  // 128 byte aligned.
  __shared__ alignas(128) int smem_buffer[SMEM_HEIGHT][SMEM_WIDTH];

  // Initialize shared memory barrier with the number of threads participating in the barrier.

  __shared__ barrier bar;




  if (threadIdx.x == 0) {
    // Initiate bulk tensor copy.
    cde::cp_async_bulk_tensor_2d_global_to_shared(&smem_buffer, &tensor_map, x, y, bar);
    // Arrive on the barrier and tell how many bytes are expected to come in.

  } else {
    // Other threads just arrive.

  }

}

int main()
{
  
  return 0;
}