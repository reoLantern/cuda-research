
#include <hip/hip_runtime.h>
#include <mma.h>
#include <iostream>
#include <iomanip>

#define WMMA_M 8
#define WMMA_N 8
#define WMMA_K 32

#define WARP_SIZE 32

#define ILP 10

int signExt4bitToInt(int num)
{ // 对符号位进行扩展
    return (num & 0x08) ? (num | 0xFFFFFFF0) : num;
}

__global__ void imma8832NaiveKernel(const uint32_t *__restrict__ A, const uint32_t *__restrict__ B,
                                    uint32_t *__restrict__ C)
{

    __shared__ uint32_t shmem_A[WMMA_M][WMMA_K / 8];
    __shared__ uint32_t shmem_B[WMMA_N][WMMA_K / 8];
    __shared__ uint32_t shmem_C[ILP][WMMA_M][WMMA_N];

    const uint32_t laneId = threadIdx.x % WARP_SIZE;

    if (laneId < 8) // 仅需要8个线程来初始化shmem
    {
        *((int4 *)(&shmem_A[laneId][0])) = *((int4 *)(&A[laneId * WMMA_K / 8]));
        *((int4 *)(&shmem_B[laneId][0])) = *((int4 *)(&B[laneId * WMMA_K / 8]));
    }

    // __syncthreads(); // 若只调用1个warp，不需要syncthreads

    uint32_t RA;
    uint32_t RB;
    uint32_t RC[2] = {0};
    uint32_t RD[2 * ILP] = {0};
    uint32_t start, stop;

    uint32_t shmem_A_lane_addr = __cvta_generic_to_shared(&shmem_A[laneId % 8][0]);
#pragma unroll
    for (int i = 0; i < ILP; i++)
    {
        asm volatile("ldmatrix.sync.aligned.m8n8.x1.shared.b16 {%0}, [%1];\n"
                     : "=r"(RA)
                     : "r"(shmem_A_lane_addr));
    }

    uint32_t shmem_B_lane_addr = __cvta_generic_to_shared(&shmem_B[laneId % 8][0]);
#pragma unroll
    for (int i = 0; i < ILP; i++)
    {
        asm volatile("ldmatrix.sync.aligned.m8n8.x1.shared.b16 {%0}, [%1];\n"
                     : "=r"(RB)
                     : "r"(shmem_B_lane_addr));
    }

    asm volatile("mov.u32 %0, %%clock;"
                 : "=r"(start)
                 :
                 : "memory");

#pragma unroll
    for (int i = 0; i < ILP; i++)
    {
        asm volatile("mma.sync.aligned.m8n8k32.row.col.s32.s4.u4.s32 {%0, %1}, {%2}, {%3}, {%4, %5};\n"
                     : "=r"(RD[i << 1]), "=r"(RD[(i << 1) + 1])
                     : "r"(RA), "r"(RB), "r"(RC[0]), "r"(RC[1]));
    }

    asm volatile("mov.u32 %0, %%clock;"
                 : "=r"(stop)
                 :
                 : "memory");

    printf("Thread%u imma starts at cycle %u, ends at cycle %u, takes %u cycles.\n", laneId, start, stop, stop - start);

    // __syncthreads(); // 若只调用1个warp，不需要syncthreads
#pragma unroll
    for (int i = 0; i < ILP; i++)
    {
        *((uint32_t *)(&shmem_C[i][laneId / 4][(laneId % 4) * 2])) = RD[i << 1];
        *((uint32_t *)(&shmem_C[i][laneId / 4][(laneId % 4) * 2] + 1)) = RD[(i << 1) + 1];
    }
    // __syncthreads(); // 若只调用1个warp，不需要syncthreads

    if (laneId < 16)
    {
        *((int4 *)(&C[laneId * 4])) = *((int4 *)(&shmem_C[0][laneId / 2][(laneId % 2) * 4]));
    }

    // __syncthreads(); // 若只调用1个warp，不需要syncthreads
}

int main()
{

    uint32_t *d_a, *h_a, *d_b, *h_b; // every 8 u4 stored in 1 uint
    uint32_t *d_c, *h_c;
    h_c = new uint32_t[WMMA_M * WMMA_N];
    h_b = new uint32_t[WMMA_K * WMMA_N / 8];
    h_a = new uint32_t[WMMA_M * WMMA_K / 8];
    hipMalloc(reinterpret_cast<void **>(&d_a), WMMA_M * WMMA_K / 8 * sizeof(uint32_t));
    hipMalloc(reinterpret_cast<void **>(&d_b), WMMA_K * WMMA_N / 8 * sizeof(uint32_t));
    hipMalloc(reinterpret_cast<void **>(&d_c), WMMA_M * WMMA_N * sizeof(int));

    for (int i = 0; i < WMMA_M * WMMA_K / 8; i++)
    {
        uint32_t value = 0;
        for (int k = 0; k < 8; k++)
        {
            value |= (((k + i) & 0x0F) << (k * 4));
        }
        h_a[i] = value; // A为row-major，每行32个int4，存放在4个int32中
    }
    for (int i = 0; i < WMMA_M * WMMA_K / 8; i++)
    {
        uint32_t value = 0;
        for (int k = 0; k < 8; k++)
        {
            value |= ((i / 2 & 0x0F) << (k * 4));
        }
        h_b[i] = value; // B为column-major，每行32个int4，存放在4个int32中
    }
    std::cout << "A=" << std::hex;
    for (int i = 0; i < WMMA_M * WMMA_K / 8; i++)
        std::cout << std::setw(8) << std::setfill('0') << h_a[i] << ",";
    std::cout << "\nB=";
    for (int i = 0; i < WMMA_M * WMMA_K / 8; i++)
        std::cout << std::setw(8) << std::setfill('0') << h_b[i] << ",";
    std::cout << std::dec << "\n";
    hipMemcpy(d_a, h_a, WMMA_M * WMMA_K / 8 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, WMMA_K * WMMA_N / 8 * sizeof(uint32_t), hipMemcpyHostToDevice);

    imma8832NaiveKernel<<<1, 32>>>(d_a, d_b, d_c);

    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, WMMA_M * WMMA_N * sizeof(uint32_t), hipMemcpyDeviceToHost);

    std::cout << "immaC=" << std::hex;
    for (int i = 0; i < WMMA_M * WMMA_N; i++)
        std::cout << h_c[i] << ",";
    std::cout << std::endl;

    /*** reference ***/
    int *ref_C = new int[WMMA_M * WMMA_N];
    for (int i = 0; i < WMMA_M; ++i) // 计算参考结果
    {
        for (int j = 0; j < WMMA_N; ++j)
        {
            int result = 0;
            for (int k = 0; k < WMMA_K; ++k)
            {
                // 获取矩阵A和B中对应的整数索引和位偏移
                int indexA = (i * WMMA_K + k) / 8;
                int offsetA = (i * WMMA_K + k) % 8;
                int indexB = (j * WMMA_K + k) / 8;
                int offsetB = (j * WMMA_K + k) % 8;
                // 从矩阵A和B中获取对应的整数值
                int valueA = h_a[indexA];
                int valueB = h_b[indexB];
                // 获取矩阵A和B中对应的4位元素值
                int elementA = signExt4bitToInt((valueA >> (offsetA * 4)) & 0x0F);
                // int elementB = signExt4bitToInt((valueB >> (offsetB * 4)) & 0x0F);
                int elementB = (valueB >> (offsetB * 4)) & 0x0F;
                // 进行乘法运算并累加结果
                result += (elementA * elementB);
            }
            ref_C[i * WMMA_N + j] = result;
        }
    }
    std::cout << "ref_C=" << std::hex;
    for (int i = 0; i < WMMA_M * WMMA_N; i++)
        std::cout << ref_C[i] << ",";
    std::cout << std::endl;
}
