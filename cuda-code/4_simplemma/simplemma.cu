
#include <hip/hip_runtime.h>
#include <mma.h>
#include <iostream>

using namespace nvcuda;

#define WMMA_M 8
#define WMMA_N 32
#define WMMA_K 16

__global__ void wmma_ker(half *a, half *b, half *c)
{
    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;

    // Initialize the output to zero
    wmma::fill_fragment(c_frag, 0.0f);

    // Load the inputs
    wmma::load_matrix_sync(a_frag, a, 16);
    wmma::load_matrix_sync(b_frag, b, 16);

    // Perform the matrix multiplication
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

    // Store the output
    wmma::store_matrix_sync(c, c_frag, 16, wmma::mem_row_major);
}

int main()
{

    half *d_a, *h_a, *d_b, *h_b;
    half *d_c, *h_c;
    h_c = new half[16 * 16];
    h_b = new half[16 * 16];
    h_a = new half[16 * 16];
    hipMalloc(&d_a, 16 * 16 * sizeof(half));
    hipMalloc(&d_b, 16 * 16 * sizeof(half));
    hipMalloc(&d_c, 16 * 16 * sizeof(float));
    for (int i = 0; i < 16 * 16; i++)
    {
        h_a[i] = 1.0f;
        h_b[i] = 1.0f;
    }
    hipMemcpy(d_a, h_a, 16 * 16 * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, 16 * 16 * sizeof(half), hipMemcpyHostToDevice);
    wmma_ker<<<1, 32>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, 16 * 16 * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 16 * 16; i++)
        std::cout << (float)h_c[i] << ",";
    std::cout << std::endl;
}
