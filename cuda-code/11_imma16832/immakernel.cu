
#include <hip/hip_runtime.h>
#include <mma.h>
#include <iostream>
#include <iomanip>

using namespace nvcuda;

#define WMMA_M 16
#define WMMA_N 8
#define WMMA_K 32

#define WARP_SIZE 32

#define ILP 10

int signExt4bitToInt(int num)
{ // 对符号位进行扩展
    return (num & 0x08) ? (num | 0xFFFFFFF0) : num;
}

__global__ void imma8832NaiveKernel(const uint32_t *__restrict__ A, const uint32_t *__restrict__ B,
                                    uint32_t *__restrict__ C)
{

    __shared__ uint32_t shmem_A[WMMA_M][WMMA_K / 8];
    __shared__ uint32_t shmem_B[WMMA_N][WMMA_K / 8];
    __shared__ uint32_t shmem_C[ILP][WMMA_M][WMMA_N];

    const uint32_t laneId = threadIdx.x % WARP_SIZE;

    if (laneId < 16) // 仅需要16个线程来初始化shmem
    {
        *((int4 *)(&shmem_A[laneId][0])) = *((int4 *)(&A[laneId * WMMA_K / 8]));
    }
    if (laneId < 8)
    {
        *((int4 *)(&shmem_B[laneId][0])) = *((int4 *)(&B[laneId * WMMA_K / 8]));
    }

    // __syncthreads(); // 若只调用1个warp，不需要syncthreads

    uint32_t RA[2];
    uint32_t RB;
    uint32_t RC[4] = {0, 0, 0, 0};
    uint32_t RD[ILP][4];

    uint32_t start, stop;

    uint32_t shmem_A_lane_addr;
    if (laneId < 8)
        shmem_A_lane_addr = __cvta_generic_to_shared(&shmem_A[laneId % 8][0]);
    else
        shmem_A_lane_addr = __cvta_generic_to_shared(&shmem_A[laneId % 8 + 8][0]);
    asm volatile("ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0, %1}, [%2];\n"
                 : "=r"(RA[0]), "=r"(RA[1])
                 : "r"(shmem_A_lane_addr));

    uint32_t shmem_B_lane_addr = __cvta_generic_to_shared(&shmem_B[laneId % 8][0]);
    asm volatile("ldmatrix.sync.aligned.m8n8.x1.shared.b16 {%0}, [%1];\n"
                 : "=r"(RB)
                 : "r"(shmem_B_lane_addr));

    asm volatile("mov.u32 %0, %%clock;"
                 : "=r"(start)
                 :
                 : "memory");

#pragma unroll
    for (int i = 0; i < ILP; i++)
    {
        asm volatile("mma.sync.aligned.m16n8k32.row.col.s32.s4.s4.s32 {%0, %1, %2, %3}, {%4, %5}, {%6}, {%7, %8, %9, %10};\n"
                     : "=r"(RD[i][0]), "=r"(RD[i][1]), "=r"(RD[i][2]), "=r"(RD[i][3])
                     : "r"(RA[0]), "r"(RA[1]), "r"(RB), "r"(RC[0]), "r"(RC[1]), "r"(RC[2]), "r"(RC[3]));
    }

    asm volatile("mov.u32 %0, %%clock;"
                 : "=r"(stop)
                 :
                 : "memory");

    printf("Thread%u imma starts at cycle %u, ends at cycle %u, takes %u cycles.\n", laneId, start, stop, stop - start);

    // __syncthreads(); // 若只调用1个warp，不需要syncthreads

#pragma unroll
    for (int i = 0; i < ILP; i++)
    {
        *((uint32_t *)(&shmem_C[i][laneId / 4][(laneId % 4) * 2])) = RD[i][0];
        *((uint32_t *)(&shmem_C[i][laneId / 4][(laneId % 4) * 2] + 1)) = RD[i][1];
        *((uint32_t *)(&shmem_C[i][laneId / 4 + 8][(laneId % 4) * 2])) = RD[i][2];
        *((uint32_t *)(&shmem_C[i][laneId / 4 + 8][(laneId % 4) * 2] + 1)) = RD[i][3];
    }
    // __syncthreads(); // 若只调用1个warp，不需要syncthreads

    *((int4 *)(&C[laneId * 4])) = *((int4 *)(&shmem_C[0][laneId / 2][(laneId % 2) * 4]));

    // __syncthreads(); // 若只调用1个warp，不需要syncthreads
}

int main()
{

    uint32_t *d_a, *h_a, *d_b, *h_b; // every 8 u4 stored in 1 uint
    uint32_t *d_c, *h_c;
    h_c = new uint32_t[WMMA_M * WMMA_N];
    h_b = new uint32_t[WMMA_K * WMMA_N / 8];
    h_a = new uint32_t[WMMA_M * WMMA_K / 8];
    hipMalloc(reinterpret_cast<void **>(&d_a), WMMA_M * WMMA_K / 8 * sizeof(uint32_t));
    hipMalloc(reinterpret_cast<void **>(&d_b), WMMA_K * WMMA_N / 8 * sizeof(uint32_t));
    hipMalloc(reinterpret_cast<void **>(&d_c), WMMA_M * WMMA_N * sizeof(int));

    for (int i = 0; i < WMMA_M * WMMA_K / 8; i++)
    {
        uint32_t value = 0;
        for (int k = 0; k < 8; k++)
        {
            value |= (((k + i) & 0x0F) << (k * 4));
        }
        h_a[i] = value; // A为row-major，每行32个int4，存放在4个int32中
    }
    for (int i = 0; i < WMMA_M * WMMA_K / 8; i++)
    {
        uint32_t value = 0;
        for (int k = 0; k < 8; k++)
        {
            value |= ((i / 2 & 0x0F) << (k * 4));
        }
        h_b[i] = value; // B为column-major，每行32个int4，存放在4个int32中
    }
    std::cout << "A=" << std::hex;
    for (int i = 0; i < WMMA_M * WMMA_K / 8; i++)
        std::cout << std::setw(8) << std::setfill('0') << h_a[i] << ",";
    std::cout << "\nB=";
    for (int i = 0; i < WMMA_M * WMMA_K / 8; i++)
        std::cout << std::setw(8) << std::setfill('0') << h_b[i] << ",";
    std::cout << std::dec << "\n";
    hipMemcpy(d_a, h_a, WMMA_M * WMMA_K / 8 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, WMMA_K * WMMA_N / 8 * sizeof(uint32_t), hipMemcpyHostToDevice);

    imma8832NaiveKernel<<<1, 32>>>(d_a, d_b, d_c);

    // cudaDeviceSynchronize();

    hipMemcpy(h_c, d_c, WMMA_M * WMMA_N * sizeof(uint32_t), hipMemcpyDeviceToHost);

    std::cout << "immaC=" << std::hex;
    for (int i = 0; i < WMMA_M * WMMA_N; i++)
        std::cout << h_c[i] << ",";
    std::cout << std::endl;

    /*** reference ***/
    int *ref_C = new int[WMMA_M * WMMA_N];
    for (int i = 0; i < WMMA_M; ++i) // 计算参考结果
    {
        for (int j = 0; j < WMMA_N; ++j)
        {
            int result = 0;
            for (int k = 0; k < WMMA_K; ++k)
            {
                // 获取矩阵A和B中对应的整数索引和位偏移
                int indexA = (i * WMMA_K + k) / 8;
                int offsetA = (i * WMMA_K + k) % 8;
                int indexB = (j * WMMA_K + k) / 8;
                int offsetB = (j * WMMA_K + k) % 8;
                // 从矩阵A和B中获取对应的整数值
                int valueA = h_a[indexA];
                int valueB = h_b[indexB];
                // 获取矩阵A和B中对应的4位元素值
                int elementA = signExt4bitToInt((valueA >> (offsetA * 4)) & 0x0F);
                // int elementB = signExt4bitToInt((valueB >> (offsetB * 4)) & 0x0F);
                int elementB = signExt4bitToInt((valueB >> (offsetB * 4)) & 0x0F);
                // 进行乘法运算并累加结果
                result += (elementA * elementB);
            }
            ref_C[i * WMMA_N + j] = result;
        }
    }
    std::cout << "ref_C=" << std::hex;
    for (int i = 0; i < WMMA_M * WMMA_N; i++)
        std::cout << ref_C[i] << ",";
    std::cout << std::endl;
}
